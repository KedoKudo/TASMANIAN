#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2017, Miroslav Stoyanov
 *
 * This file is part of
 * Toolkit for Adaptive Stochastic Modeling And Non-Intrusive ApproximatioN: TASMANIAN
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice, this list of conditions
 *    and the following disclaimer in the documentation and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its contributors may be used to endorse
 *    or promote products derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES,
 * INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED.
 * IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY,
 * OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA,
 * OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 * UT-BATTELLE, LLC AND THE UNITED STATES GOVERNMENT MAKE NO REPRESENTATIONS AND DISCLAIM ALL WARRANTIES, BOTH EXPRESSED AND IMPLIED.
 * THERE ARE NO EXPRESS OR IMPLIED WARRANTIES OF MERCHANTABILITY OR FITNESS FOR A PARTICULAR PURPOSE, OR THAT THE USE OF THE SOFTWARE WILL NOT INFRINGE ANY PATENT,
 * COPYRIGHT, TRADEMARK, OR OTHER PROPRIETARY RIGHTS, OR THAT THE SOFTWARE WILL ACCOMPLISH THE INTENDED RESULTS OR THAT THE SOFTWARE OR ITS USE WILL NOT RESULT IN INJURY OR DAMAGE.
 * THE USER ASSUMES RESPONSIBILITY FOR ALL LIABILITIES, PENALTIES, FINES, CLAIMS, CAUSES OF ACTION, AND COSTS AND EXPENSES, CAUSED BY, RESULTING FROM OR ARISING OUT OF,
 * IN WHOLE OR IN PART THE USE, STORAGE OR DISPOSAL OF THE SOFTWARE.
 */

#ifndef __TASMANIAN_SPARSE_GRID_CUDA_KERNELS_CU
#define __TASMANIAN_SPARSE_GRID_CUDA_KERNELS_CU

#include "tsgAcceleratedDataStructures.hpp"

#include "tsgCudaMacros.hpp"

#include "tsgCudaLinearAlgebra.hpp"

#include "tsgCudaBasisEvaluations.hpp"

// several kernels assume linear distribution the threads and can be executed with "practically unlimited" number of threads
// thus we can set this to the CUDA max number of threads, based on the computer version
#define _MAX_CUDA_THREADS 1024

namespace TasGrid{

void TasCUDA::dtrans2can(bool use01, int dims, int num_x, int pad_size, const double *gpu_trans_a, const double *gpu_trans_b, const double *gpu_x_transformed, double *gpu_x_canonical){
    int num_blocks = (num_x * dims) / _MAX_CUDA_THREADS + (((num_x * dims) % _MAX_CUDA_THREADS == 0) ? 0 : 1);
    if (num_blocks >= 65536) num_blocks = 65536;
    tasgpu_transformed_to_canonical<double, double, _MAX_CUDA_THREADS><<<num_blocks, _MAX_CUDA_THREADS, (2*pad_size) * sizeof(double)>>>(dims, num_x, pad_size, gpu_trans_a, gpu_trans_b, gpu_x_transformed, gpu_x_canonical);
    if (use01) tasgpu_m11_to_01<double, _MAX_CUDA_THREADS><<<num_blocks, _MAX_CUDA_THREADS>>>(dims * num_x, gpu_x_canonical);
}

// local polynomial basis functions, DENSE algorithm
void TasCUDA::devalpwpoly(int order, TypeOneDRule rule, int dims, int num_x, int num_points, const double *gpu_x, const double *gpu_nodes, const double *gpu_support, double *gpu_y){
    // each block thread runs 1024 threads and processes 32 points (or basis functions)
    int num_blocks = (num_points / 32) + ((num_points % 32 == 0) ? 0 : 1);
    // order == 1 is considered "default" so that the compiler doesn't complain about missing default statement
    // semilocalp cannot have order less than 2, only rule_localp can have order 0 (this gets overwrittein in makeLocalPolynomialGrid())
    if (rule == rule_localp){
        switch(order){
            case 0:
                    tasgpu_devalpwpoly<double, 0, rule_localp, 32, 64><<<num_blocks, 1024>>>(dims, num_x, num_points, gpu_x, gpu_nodes, gpu_support, gpu_y);
                    break;
            case 2: tasgpu_devalpwpoly<double, 2, rule_localp, 32, 64><<<num_blocks, 1024>>>(dims, num_x, num_points, gpu_x, gpu_nodes, gpu_support, gpu_y);
                    break;
            default:
                    tasgpu_devalpwpoly<double, 1, rule_localp, 32, 64><<<num_blocks, 1024>>>(dims, num_x, num_points, gpu_x, gpu_nodes, gpu_support, gpu_y);
        }
    }else if (rule == rule_localp0){
        switch(order){
            case 2: tasgpu_devalpwpoly<double, 2, rule_localp0, 32, 64><<<num_blocks, 1024>>>(dims, num_x, num_points, gpu_x, gpu_nodes, gpu_support, gpu_y);
                    break;
            default:
                    tasgpu_devalpwpoly<double, 1, rule_localp0, 32, 64><<<num_blocks, 1024>>>(dims, num_x, num_points, gpu_x, gpu_nodes, gpu_support, gpu_y);
        }
    }else if (rule == rule_localpb){
        switch(order){
            case 2: tasgpu_devalpwpoly<double, 2, rule_localpb, 32, 64><<<num_blocks, 1024>>>(dims, num_x, num_points, gpu_x, gpu_nodes, gpu_support, gpu_y);
                    break;
            default:
                    tasgpu_devalpwpoly<double, 1, rule_localpb, 32, 64><<<num_blocks, 1024>>>(dims, num_x, num_points, gpu_x, gpu_nodes, gpu_support, gpu_y);
        }
    }else{ // rule == rule_semilocalp
        tasgpu_devalpwpoly<double, 2, rule_semilocalp, 32, 64><<<num_blocks, 1024>>>(dims, num_x, num_points, gpu_x, gpu_nodes, gpu_support, gpu_y);
    }
}

// there is a switch statement that realizes templates for each combination of rule/order
// make one function that covers that switch, the rest is passed from devalpwpoly_sparse
template<typename T, int THREADS, int TOPLEVEL, bool fill>
inline void devalpwpoly_sparse_realize_rule_order(int order, TypeOneDRule rule,
                                          int dims, int num_x, int num_points,
                                          const T *x, const T *nodes, const T *support,
                                          const int *hpntr, const int *hindx, int num_roots, const int *roots,
                                          int *spntr, int *sindx, T *svals){
    int num_blocks = num_x / THREADS + ((num_x % THREADS == 0) ? 0 : 1);
    if (num_blocks >= 65536) num_blocks = 65536;
    if (rule == rule_localp){
        switch(order){
            case 0:
                tasgpu_devalpwpoly_sparse<T, THREADS, TOPLEVEL, 0, rule_localp, fill><<<num_blocks, THREADS>>>
                    (dims, num_x, num_points, x, nodes, support, hpntr, hindx, num_roots, roots, spntr, sindx, svals);
                break;
            case 2:
                tasgpu_devalpwpoly_sparse<T, THREADS, TOPLEVEL, 2, rule_localp, fill><<<num_blocks, THREADS>>>
                    (dims, num_x, num_points, x, nodes, support, hpntr, hindx, num_roots, roots, spntr, sindx, svals);
                break;
            default:
                tasgpu_devalpwpoly_sparse<T, THREADS, TOPLEVEL, 1, rule_localp, fill><<<num_blocks, THREADS>>>
                    (dims, num_x, num_points, x, nodes, support, hpntr, hindx, num_roots, roots, spntr, sindx, svals);
        }
    }else if (rule == rule_localp0){
        switch(order){
            case 2:
                tasgpu_devalpwpoly_sparse<T, THREADS, TOPLEVEL, 2, rule_localp0, fill><<<num_blocks, THREADS>>>
                    (dims, num_x, num_points, x, nodes, support, hpntr, hindx, num_roots, roots, spntr, sindx, svals);
                break;
            default:
                tasgpu_devalpwpoly_sparse<T, THREADS, TOPLEVEL, 1, rule_localp0, fill><<<num_blocks, THREADS>>>
                    (dims, num_x, num_points, x, nodes, support, hpntr, hindx, num_roots, roots, spntr, sindx, svals);
        }
    }else if (rule == rule_localpb){
        switch(order){
            case 2:
                tasgpu_devalpwpoly_sparse<T, THREADS, TOPLEVEL, 2, rule_localpb, fill><<<num_blocks, THREADS>>>
                    (dims, num_x, num_points, x, nodes, support, hpntr, hindx, num_roots, roots, spntr, sindx, svals);
                break;
            default:
                tasgpu_devalpwpoly_sparse<T, THREADS, TOPLEVEL, 1, rule_localpb, fill><<<num_blocks, THREADS>>>
                    (dims, num_x, num_points, x, nodes, support, hpntr, hindx, num_roots, roots, spntr, sindx, svals);
        }
    }else{ // rule == rule_semilocalp
        tasgpu_devalpwpoly_sparse<T, THREADS, TOPLEVEL, 2, rule_semilocalp, fill><<<num_blocks, THREADS>>>
            (dims, num_x, num_points, x, nodes, support, hpntr, hindx, num_roots, roots, spntr, sindx, svals);
    }
}

// local polynomial basis functions, SPARSE algorithm (2 passes, one pass to compue the non-zeros and one pass to evaluate)
void TasCUDA::devalpwpoly_sparse(int order, TypeOneDRule rule, int dims, int num_x, int num_points, const double *gpu_x, const cudaDoubles &gpu_nodes, const cudaDoubles &gpu_support,
                            const cudaInts &gpu_hpntr, const cudaInts &gpu_hindx, const  cudaInts &gpu_roots, cudaInts &gpu_spntr, cudaInts &gpu_sindx, cudaDoubles &gpu_svals){
    gpu_spntr.resize(num_x + 1);
    // call with fill == false to count the non-zeros per row of the matrix
    devalpwpoly_sparse_realize_rule_order<double, 64, 46, false>
        (order, rule, dims, num_x, num_points, gpu_x, gpu_nodes.data(), gpu_support.data(), gpu_hpntr.data(), gpu_hindx.data(), (int) gpu_roots.size(), gpu_roots.data(), gpu_spntr.data(), 0, 0);

    std::vector<int> cpu_spntr;
    gpu_spntr.unload(cpu_spntr);
    cpu_spntr[0] = 0;
    int nz = 0;
    for(auto &i : cpu_spntr){
        i += nz;
        nz = i;
    }
    gpu_spntr.load(cpu_spntr);
    gpu_sindx.resize(nz);
    gpu_svals.resize(nz);
    // call with fill == true to load the non-zeros
    devalpwpoly_sparse_realize_rule_order<double, 64, 46, true>
        (order, rule, dims, num_x, num_points, gpu_x, gpu_nodes.data(), gpu_support.data(), gpu_hpntr.data(), gpu_hindx.data(), (int) gpu_roots.size(), gpu_roots.data(), gpu_spntr.data(), gpu_sindx.data(), gpu_svals.data());
}

// Sequence Grid basis evaluations
void TasCUDA::devalseq(int dims, int num_x, const std::vector<int> &max_levels, const double *gpu_x, const cudaInts &num_nodes, const cudaInts &points, const cudaDoubles &nodes, const cudaDoubles &coeffs, double *gpu_result){
    std::vector<int> offsets(dims);
    offsets[0] = 0;
    for(int d=1; d<dims; d++) offsets[d] = offsets[d-1] + num_x * (max_levels[d-1] + 1);
    size_t num_total = offsets[dims-1] + num_x * (max_levels[dims-1] + 1);

    int maxl = max_levels[0]; for(auto l : max_levels) if (maxl < l) maxl = l;

    cudaInts gpu_offsets(offsets);
    cudaDoubles cache1D(num_total);
    int num_blocks = num_x / _MAX_CUDA_THREADS + ((num_x % _MAX_CUDA_THREADS == 0) ? 0 : 1);

    tasgpu_dseq_build_cache<double, _MAX_CUDA_THREADS><<<num_blocks, _MAX_CUDA_THREADS>>>
        (dims, num_x, gpu_x, nodes.data(), coeffs.data(), maxl+1, gpu_offsets.data(), num_nodes.data(), cache1D.data());

    num_blocks = num_x / 32 + ((num_x % 32 == 0) ? 0 : 1);
    tasgpu_dseq_eval_sharedpoints<double, 32><<<num_blocks, 1024>>>
        (dims, num_x, (int) points.size() / dims, points.data(), gpu_offsets.data(), cache1D.data(), gpu_result);
}

// Fourier Grid basis evaluations
void TasCUDA::devalfor(int dims, int num_x, const std::vector<int> &max_levels, const double *gpu_x, const cudaInts &num_nodes, const cudaInts &points, double *gpu_wreal, double *gpu_wimag){
    std::vector<int> max_nodes(dims);
    for(int j=0; j<dims; j++){
        int n = 1;
        for(int i=0; i<max_levels[j]; i++) n *= 3;
        max_nodes[j] = n;
    }

    std::vector<int> offsets(dims);
    offsets[0] = 0;
    for(int d=1; d<dims; d++) offsets[d] = offsets[d-1] + 2 * num_x * (max_nodes[d-1] + 1);
    size_t num_total = offsets[dims-1] + 2 * num_x * (max_nodes[dims-1] + 1);

    cudaInts gpu_offsets(offsets);
    cudaDoubles cache1D(num_total);
    int num_blocks = num_x / _MAX_CUDA_THREADS + ((num_x % _MAX_CUDA_THREADS == 0) ? 0 : 1);

    tasgpu_dfor_build_cache<double, _MAX_CUDA_THREADS><<<num_blocks, _MAX_CUDA_THREADS>>>
        (dims, num_x, gpu_x, gpu_offsets.data(), num_nodes.data(), cache1D.data());

    num_blocks = num_x / 32 + ((num_x % 32 == 0) ? 0 : 1);
    if (gpu_wimag == 0){
        tasgpu_dfor_eval_sharedpoints<double, 32, true><<<num_blocks, 1024>>>
            (dims, num_x, (int) points.size() / dims, points.data(), gpu_offsets.data(), cache1D.data(), gpu_wreal, 0);
    }else{
        tasgpu_dfor_eval_sharedpoints<double, 32, false><<<num_blocks, 1024>>>
            (dims, num_x, (int) points.size() / dims, points.data(), gpu_offsets.data(), cache1D.data(), gpu_wreal, gpu_wimag);
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//       Linear Algebra
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
#ifdef __TASMANIAN_COMPILE_FALLBACK_CUDA_KERNELS__
void TasCUDA::cudaDgemm(int M, int N, int K, const double *gpu_a, const double *gpu_b, double *gpu_c){ // gpu_c = gpu_a * gpu_b, gpu_c is M by N
    int blocks = (N / 96) + (((N % 96) == 0) ? 0 : 1);
    blocks *= (M / 96) + (((M % 96) == 0) ? 0 : 1);
    while(blocks > 65536) blocks = 65536;
    tasgpu_cudaTgemm<double, 32, 96><<<blocks, 1024>>>(M, N, K, gpu_a, gpu_b, gpu_c);
}

void TasCUDA::cudaSparseMatmul(int M, int N, int num_nz, const int* gpu_spntr, const int* gpu_sindx, const double* gpu_svals, const double *gpu_B, double *gpu_C){
    int blocks = M / 64 + ((M % 64 == 0) ? 0 : 1);
    tasgpu_sparse_matmul<double, 64><<<blocks, 64>>>(M, N, num_nz, gpu_spntr, gpu_sindx, gpu_svals, gpu_B, gpu_C);
}

void TasCUDA::cudaSparseVecDenseMat(int M, int N, int num_nz, const double *A, const int *indx, const double *vals, double *C){
    int num_blocks = N / _MAX_CUDA_THREADS + ((N % _MAX_CUDA_THREADS == 0) ? 0 : 1);
    if (num_blocks< 65536){
        tasgpu_sparse_matveci<double, _MAX_CUDA_THREADS, 1><<<num_blocks, _MAX_CUDA_THREADS>>>(M, N, num_nz, A, indx, vals, C);
    }else{
        num_blocks = N / (2 * _MAX_CUDA_THREADS) + ((N % (2 * _MAX_CUDA_THREADS) == 0) ? 0 : 1);
        if (num_blocks< 65536){
            tasgpu_sparse_matveci<double, _MAX_CUDA_THREADS, 2><<<num_blocks, _MAX_CUDA_THREADS>>>(M, N, num_nz, A, indx, vals, C);
        }else{
            num_blocks = N / (3 * _MAX_CUDA_THREADS) + ((N % (3 * _MAX_CUDA_THREADS) == 0) ? 0 : 1);
            if (num_blocks< 65536){
                tasgpu_sparse_matveci<double, _MAX_CUDA_THREADS, 3><<<num_blocks, _MAX_CUDA_THREADS>>>(M, N, num_nz, A, indx, vals, C);
            }
        }
    }
}

void TasCUDA::convert_sparse_to_dense(int num_rows, int num_columns, const int *pntr, const int *indx, const double *vals, double *destination){
    int n = num_rows * num_columns;
    int num_blocks = n / _MAX_CUDA_THREADS + ((n % _MAX_CUDA_THREADS == 0) ? 0 : 1);
    if (num_blocks >= 65536) num_blocks = 65536;
    tascuda_fill<double, _MAX_CUDA_THREADS><<<num_blocks, _MAX_CUDA_THREADS>>>(n, 0.0, destination);
    num_blocks = num_rows;
    if (num_blocks >= 65536) num_blocks = 65536;
    tascuda_sparse_to_dense<double, 64><<<num_blocks, 64>>>(num_rows, num_columns, pntr, indx, vals, destination);
}
#endif

}

#endif
